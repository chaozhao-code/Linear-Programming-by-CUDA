#include "hip/hip_runtime.h"
#include "floating_number_helper.h"
#include "input_output.h"
#include <float.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include <iostream>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\transform.h>
//
// this function is what you need to finish
// @Usage : to solve the problem
// @Input : input containing all data needed
// @Output: answer containing all necessary data
//  you can find the definition of the two structs above in
//      input_output.h
//

answer * compute(inputs * input) {
	int num = input->number;
	line ** lines = input->lines;
	answer * ans = (answer *)malloc(sizeof(answer));
	double tmp = 0;
	//double min = MAXFLOAT;
	double min = FLT_MAX;
	for (int i = 0; i < num - 1; i++) {
		line * now_line = lines[i];
		for (int j = i + 1; j < num; j++) {
			line * tmp_line = lines[j];
			point * tmp_node = generate_intersection_point(now_line, tmp_line);
			tmp = input->obj_function_param_a * tmp_node->pos_x + input->obj_function_param_b * tmp_node->pos_y;
			if (tmp <= min) {
				for (int k = 0; k < num; k++) {
					if ((tmp_node->pos_x * lines[k]->param_a + tmp_node->pos_y * lines[k]->param_b) < lines[k]->param_c) {
						break;
					}
					else if (k == num - 1) {
						min = tmp;
						ans->answer_b = tmp;
						ans->intersection_point = tmp_node;
						ans->line1 = now_line;
						ans->line2 = tmp_line;
					}
				}
			}
		}
	}
	return ans;
}

__device__ int equals_gpu(double num1, double num2) {
	return fabs(num1 - num2) < EPS ? TRUE : FALSE;
}

__device__ int is_parallel_gpu(line * line1, line * line2) {
	return equals_gpu(line1->param_a * line2->param_b, line1->param_b * line2->param_a);
}

__device__ point * generate_intersection_point_gpu(line * line1, line * line2) {
	if (is_parallel_gpu(line1, line2)) {
		return NULL;
	}
	point * new_point = (point *)malloc(sizeof(point));
	new_point->pos_x = (line1->param_c * line2->param_b - line1->param_b * line2->param_c)
		/ (line1->param_a * line2->param_b - line1->param_b * line2->param_a);
	new_point->pos_y = (line1->param_c * line2->param_a - line1->param_a * line2->param_c)
		/ (line1->param_b * line2->param_a - line1->param_a * line2->param_b);
	return new_point;
}

__global__ static void gpu_compute(int * dev_num, double * dev_param_a, double * dev_param_b, line * dev_line_array, double * valmax) {
	int num = *dev_num;
	double a = *dev_param_a;
	double b = *dev_param_b;
	int i = threadIdx.x;
	for (int j = i + 1; j < num; j++) 
	{
		point * tmp_node = generate_intersection_point_gpu(&dev_line_array[i], &dev_line_array[j]);
		for (int k = 0; k < num; k++) 
		{
			double tmp = a * tmp_node->pos_x + b * tmp_node->pos_y;
			if ((tmp_node->pos_x * dev_line_array[k].param_a + tmp_node->pos_y * dev_line_array[k].param_b) < dev_line_array[k].param_c - 0.000001)
			{
				//printf("%f", tmp);
				break;
			}
			else if (k == num - 1)
			{
				valmax[i * num + j] = tmp;
			}
		}
	}
}

/*
int main() {
	// 1. get the input data
	inputs * input = read_from_file("C:/Users/hzauz/Desktop/sws3003_assignment 1/test_cases/100_0.dat");
	int num = input->number;
	printf("%d\n", num);
	// 2. transform data from CPU to GPU
	double * valmax = (double*)malloc(sizeof(double) * num * num);                      // for output
	for (int i = 0; i < num * num; i++)
	{
		valmax[i] = FLT_MAX;
	}
	double * dev_valmax;
	hipMalloc((void**)&dev_valmax, sizeof(double) * num * num);
	hipMemcpy(dev_valmax, valmax, sizeof(double) * num * num, hipMemcpyHostToDevice);
	
	line * line_array = (line*)malloc(sizeof(line) * num);                              // for input line data
	for (int i = 0; i < num; i++)
	{
		line_array[i] = * ((input->lines)[i]);
	}
	line * dev_line_array;
	hipMalloc((void**)&dev_line_array, sizeof(line) * num);
	hipMemcpy(dev_line_array, line_array, sizeof(line) * num, hipMemcpyHostToDevice);
	
	int * dev_num;                                                                      // for input the number of line
	hipMalloc((void**)&dev_num, sizeof(int));
	hipMemcpy(dev_num, &num, sizeof(int), hipMemcpyHostToDevice);

	double * dev_param_a;                                                               // for input the number of a and b
	double param_a = input->obj_function_param_a;
	hipMalloc((void**)&dev_param_a, sizeof(double));
	hipMemcpy(dev_param_a, &param_a, sizeof(double), hipMemcpyHostToDevice);
	double * dev_param_b;
	double param_b = input->obj_function_param_b;
	hipMalloc((void**)&dev_param_b, sizeof(double));
	hipMemcpy(dev_param_b, &param_b, sizeof(double), hipMemcpyHostToDevice);
	// 3.computing by using gpu
	gpu_compute<<<1,num>>>(dev_num, dev_param_a, dev_param_b, dev_line_array, dev_valmax);

	// 4.getting the final answer
	hipMemcpy(valmax,dev_valmax, sizeof(double*) * num * num, hipMemcpyDeviceToHost);
	int minidx_x = 0;
	int minidx_y = 0;
	double minval = FLT_MAX;
	for (int i = 0; i < num * num; i++)
	{	
		if (valmax[i] < minval)
		{
			minidx_x = i / num;
			minidx_y = i - minidx_x * num;
			minval = valmax[i];
		}
	}
	printf("minidx_x:%d, minidy_y:%d", minidx_x, minidx_y);
	answer * ans = (answer *)malloc(sizeof(answer));
	point * tmp_node = generate_intersection_point(input->lines[minidx_x], input->lines[minidx_y]);
	double tmp = input->obj_function_param_a * tmp_node->pos_x + input->obj_function_param_b * tmp_node->pos_y;
	ans->answer_b = tmp;
	ans->intersection_point = tmp_node;
	ans->line1 = input->lines[minidx_x];
	ans->line2 = input->lines[minidx_y];
	// 3. display result and free memory
	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	free_inputs(&input);
	hipFree(dev_line_array);
	free(line_array);
	free_ans(&ans);
	free(ans_string);
	hipFree(dev_valmax);
	free(valmax);

	return 0;
}
*/
struct rotate_functor
{
	const double a;
	const double b;

	rotate_functor(double _a, double _b) : a(_a), b(_b) {}

	__host__ __device__
		line operator() (line&rotate_line)
	{
		double x, y, z;
		x = rotate_line.param_a;
		y = rotate_line.param_b;
		z = rotate_line.param_c;
		bool big_0_before_rotate = z > 0;
		rotating(x, y, z);
		bool big_0_after_rotate = z > 0;
		if (big_0_before_rotate != big_0_after_rotate)
		{
			x = -x;
			y = -y;
			z = -z;
		}
		rotate_line.param_a = x;
		rotate_line.param_b = y;
		rotate_line.param_c = z;
		return rotate_line;
	}

	__host__ __device__
		void rotating(double& u, double& v, double& w)
	{
		double point_one_x, point_one_y, point_two_x, point_two_y = 0;
		point_one_x = (a * w) / u;
		point_one_y = (b * w) / u;
		point_two_x = (b * w) / v;
		point_two_y = ((-a) * w) / v;
		get_position(point_one_x, point_one_y);
		get_position(point_two_x, point_two_y);
		u = point_one_y - point_two_y;
		v = -(point_one_x - point_two_x);
		w = u * point_two_x + v * point_two_y;
	}

	__host__ __device__
		void get_position(double& pos_x, double& pos_y)
	{
		if (pos_x < 0)
		{
			pos_x = -(sqrt(abs(pos_x)));
		}
		else
			pos_x = sqrt(pos_x);
		if (pos_y < 0)
		{
			pos_y = -(sqrt(abs(pos_y)));
		}
		else
			pos_y = sqrt(pos_y);
	}

};

int main()
{
	inputs * input = read_from_file(""); //replace this with your input file
	int num = input->number;
	//line ** lines = input->lines;
	//thrust::device_vector<line> dev_lines(num);
	//for (int i = 0; i < num; i++)
	//{
	//	dev_lines[i] = (*lines)[i];
	//}
	for (int i = 0; i < num; i++)
	{
		std::cout << (*input->lines)[i].param_a << "\n" << std::endl;
	}
	double a, b;
	a = input->obj_function_param_a;
	b = input->obj_function_param_b;
	//thrust::transform(dev_lines.begin(), dev_lines.end(), dev_lines.begin(), rotate_functor(a, b));	
	//for (int i = 0; i < num; i++)
	//{
	//	std::cout << dev_lines[i].param_a << "\n" << std::endl;
	//}
	return 0;
}