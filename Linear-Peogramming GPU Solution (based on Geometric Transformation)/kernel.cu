#include "hip/hip_runtime.h"
#include "floating_number_helper.h"
#include "input_output.h"
#include <float.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include <iostream>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\transform.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>
#include <thrust/partition.h>
#include <thrust/extrema.h>
#include <thrust/remove.h>

//
// this function is what you need to finish
// @Usage : to solve the problem
// @Input : input containing all data needed
// @Output: answer containing all necessary data
//  you can find the definition of the two structs above in
//      input_output.h
//

answer * compute(inputs * input) {
	int num = input->number;
	line ** lines = input->lines;
	answer * ans = (answer *)malloc(sizeof(answer));
	double tmp = 0;
	//double min = MAXFLOAT;
	double min = FLT_MAX;
	for (int i = 0; i < num - 1; i++) {
		line * now_line = lines[i];
		for (int j = i + 1; j < num; j++) {
			line * tmp_line = lines[j];
			point * tmp_node = generate_intersection_point(now_line, tmp_line);
			//std::cout << tmp_node->pos_x << "   " << tmp_node->pos_y << std::endl;
			tmp = input->obj_function_param_a * tmp_node->pos_x + input->obj_function_param_b * tmp_node->pos_y;
			//std::cout << tmp << std::endl;
			if (tmp <= min) {
				for (int k = 0; k < num; k++) {
					if ((tmp_node->pos_x * lines[k]->param_a + tmp_node->pos_y * lines[k]->param_b) < lines[k]->param_c) {
						break;
					}
					else if (k == num - 1) {
						//std::cout << "we can get answer" << std::endl;
						min = tmp;
						ans->answer_b = tmp;
						ans->intersection_point = tmp_node;
						ans->line1 = now_line;
						ans->line2 = tmp_line;
					}
				}
			}
		}
	}
	return ans;
}

__device__ int equals_gpu(double num1, double num2) {
	return fabs(num1 - num2) < EPS ? TRUE : FALSE;
}

__device__ int is_parallel_gpu(line * line1, line * line2) {
	return equals_gpu(line1->param_a * line2->param_b, line1->param_b * line2->param_a);
}

__device__ point * generate_intersection_point_gpu(line * line1, line * line2) {
	if (is_parallel_gpu(line1, line2)) {
		return NULL;
	}
	point * new_point = (point *)malloc(sizeof(point));
	new_point->pos_x = (line1->param_c * line2->param_b - line1->param_b * line2->param_c)
		/ (line1->param_a * line2->param_b - line1->param_b * line2->param_a);
	new_point->pos_y = (line1->param_c * line2->param_a - line1->param_a * line2->param_c)
		/ (line1->param_b * line2->param_a - line1->param_a * line2->param_b);
	return new_point;
}

__global__ static void gpu_compute(int * dev_num, double * dev_param_a, double * dev_param_b, line * dev_line_array, double * valmax) {
	int num = *dev_num;
	double a = *dev_param_a;
	double b = *dev_param_b;
	int i = threadIdx.x;
	for (int j = i + 1; j < num; j++) 
	{
		point * tmp_node = generate_intersection_point_gpu(&dev_line_array[i], &dev_line_array[j]);
		for (int k = 0; k < num; k++) 
		{
			double tmp = a * tmp_node->pos_x + b * tmp_node->pos_y;
			if ((tmp_node->pos_x * dev_line_array[k].param_a + tmp_node->pos_y * dev_line_array[k].param_b) < dev_line_array[k].param_c - 0.000001)
			{
				//printf("%f", tmp);
				break;
			}
			else if (k == num - 1)
			{
				valmax[i * num + j] = tmp;
			}
		}
	}
}

/*
int main() {
	// 1. get the input data
	inputs * input = read_from_file("C:/Users/hzauz/Desktop/sws3003_assignment 1/test_cases/100_0.dat");
	int num = input->number;
	printf("%d\n", num);
	// 2. transform data from CPU to GPU
	double * valmax = (double*)malloc(sizeof(double) * num * num);                      // for output
	for (int i = 0; i < num * num; i++)
	{
		valmax[i] = FLT_MAX;
	}
	double * dev_valmax;
	hipMalloc((void**)&dev_valmax, sizeof(double) * num * num);
	hipMemcpy(dev_valmax, valmax, sizeof(double) * num * num, hipMemcpyHostToDevice);
	
	line * line_array = (line*)malloc(sizeof(line) * num);                              // for input line data
	for (int i = 0; i < num; i++)
	{
		line_array[i] = * ((input->lines)[i]);
	}
	line * dev_line_array;
	hipMalloc((void**)&dev_line_array, sizeof(line) * num);
	hipMemcpy(dev_line_array, line_array, sizeof(line) * num, hipMemcpyHostToDevice);
	
	int * dev_num;                                                                      // for input the number of line
	hipMalloc((void**)&dev_num, sizeof(int));
	hipMemcpy(dev_num, &num, sizeof(int), hipMemcpyHostToDevice);

	double * dev_param_a;                                                               // for input the number of a and b
	double param_a = input->obj_function_param_a;
	hipMalloc((void**)&dev_param_a, sizeof(double));
	hipMemcpy(dev_param_a, &param_a, sizeof(double), hipMemcpyHostToDevice);
	double * dev_param_b;
	double param_b = input->obj_function_param_b;
	hipMalloc((void**)&dev_param_b, sizeof(double));
	hipMemcpy(dev_param_b, &param_b, sizeof(double), hipMemcpyHostToDevice);
	// 3.computing by using gpu
	gpu_compute<<<1,num>>>(dev_num, dev_param_a, dev_param_b, dev_line_array, dev_valmax);

	// 4.getting the final answer
	hipMemcpy(valmax,dev_valmax, sizeof(double*) * num * num, hipMemcpyDeviceToHost);
	int minidx_x = 0;
	int minidx_y = 0;
	double minval = FLT_MAX;
	for (int i = 0; i < num * num; i++)
	{	
		if (valmax[i] < minval)
		{
			minidx_x = i / num;
			minidx_y = i - minidx_x * num;
			minval = valmax[i];
		}
	}
	printf("minidx_x:%d, minidy_y:%d", minidx_x, minidx_y);
	answer * ans = (answer *)malloc(sizeof(answer));
	point * tmp_node = generate_intersection_point(input->lines[minidx_x], input->lines[minidx_y]);
	double tmp = input->obj_function_param_a * tmp_node->pos_x + input->obj_function_param_b * tmp_node->pos_y;
	ans->answer_b = tmp;
	ans->intersection_point = tmp_node;
	ans->line1 = input->lines[minidx_x];
	ans->line2 = input->lines[minidx_y];
	// 3. display result and free memory
	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	free_inputs(&input);
	hipFree(dev_line_array);
	free(line_array);
	free_ans(&ans);
	free(ans_string);
	hipFree(dev_valmax);
	free(valmax);

	return 0;
}
*/
struct rotate_functor
{
	const double a;
	const double b;

	rotate_functor(double _a, double _b) : a(_a), b(_b) {}

	__host__ __device__
		line operator() (line&rotate_line)
	{
		double x, y, z;
		line copy_line = rotate_line;
		x = copy_line.param_a;
		y = copy_line.param_b;
		z = copy_line.param_c;
		bool big_0_before_rotate = z > 0;
		rotating(x, y, z);
		bool big_0_after_rotate = z > 0;
		if (big_0_before_rotate != big_0_after_rotate)
		{
			x = -x;
			y = -y;
			z = -z;
		}
		rotate_line.param_a = x;
		rotate_line.param_b = y;
		rotate_line.param_c = z;
		if (rotate_line.param_b < 0)
		{
			rotate_line.I_value = I_minus;
		}
		else if (rotate_line.param_b > 0)
		{
			rotate_line.I_value = I_plus;
		}
		else 
			rotate_line.I_value = I_0;
		rotate_line.slope_value = -(x / y);
		return rotate_line;
	}

	__host__ __device__
		void rotating(double& u, double& v, double& w)
	{
		double point_one_x, point_one_y, point_two_x, point_two_y = 0;
		point_one_y = (a * w) / u;
		point_one_x = (b * w) / u;
		point_two_y = (b * w) / v;
		point_two_x = ((-a) * w) / v;
		get_position(point_one_x, point_one_y);
		get_position(point_two_x, point_two_y);
		u = point_one_y - point_two_y;
		v = point_two_x - point_one_x;
		w = point_one_y * point_two_x - point_two_y * point_one_x; 
	}

	__host__ __device__
		void get_position(double& pos_x, double& pos_y)
	{
		double obj_length = sqrt(a*a + b*b);
		pos_x = pos_x / obj_length;
		pos_y = pos_y / obj_length;

	}

};

/*
struct I_functor
{
	const double a;

	I_functor(double _a) : a(_a) {}
	__host__ __device__
		I operator() (line&anyline)
	{
		if (anyline.param_b > 0)
			return I_pos;
		else if (anyline.param_b < 0)
			return I_neg;
		else
			return I_0;
	}
};
*/

struct generate_intersection
{
	const double x;
	generate_intersection(double _x) : x(_x) {}
	__host__ __device__
		line operator() (line&anyline)
	{
		line tmp_line = anyline;
		tmp_line.distance_with_testline = (tmp_line.param_c - tmp_line.param_a * x) / tmp_line.param_b;
		return tmp_line;
	}
};

void rotate_all(inputs*& input)
{
	line** lines = input->lines;
	int num = input->number;
	double pivot_a = input->obj_function_param_a;
	double pivot_b = input->obj_function_param_b;
	input->obj_function_param_a = 0;
	input->obj_function_param_b = sqrt(pivot_a * pivot_a + pivot_b * pivot_b);
	double cosine_value = pivot_b / (sqrt(pivot_a * pivot_a + pivot_b * pivot_b));
	double sine_value = pivot_a / (sqrt(pivot_a * pivot_a + pivot_b * pivot_b));
	for (int i = 0; i < num; ++i) {
		double new_a = (lines[i]->param_a * cosine_value - lines[i]->param_b * sine_value);
		double new_b = (lines[i]->param_a * sine_value + lines[i]->param_b * cosine_value);
		line* new_line = generate_line_from_abc(new_a, new_b, lines[i]->param_c);
		lines[i] = new_line;
	}
}

struct is_I_minus
{
	__host__ __device__
		bool operator()(line anyline)
	{
		return (anyline.I_value == -1);
	}
};

struct compare_by_distance
{
	__host__ __device__
		bool operator()(line line_one, line line_two)
	{
		return line_one.distance_with_testline < line_two.distance_with_testline;
	}
};

struct is_move
{
	const double slope;
	const point max_intersection;
	is_move(double _slope, point _max_intersection) : slope(_slope), max_intersection(_max_intersection) {}
	__host__ __device__
		bool operator()(line anyline)
	{
		if (anyline.slope_value + 0.0000001 < slope)   // + 0.001 ��Ϊ�˷�ֹ��������
			return true;
		else if (max_intersection.pos_y > (((anyline.param_c - anyline.param_a * max_intersection.pos_x) / anyline.param_b) + 0.0000001))
			return true;
		else
			return false;
	}
};

struct is_move_for_count
{
	const double slope;
	const point max_intersection;
	is_move_for_count(double _slope, point _max_intersection) : slope(_slope), max_intersection(_max_intersection) {}
	__host__ __device__
		bool operator()(line anyline)
	{
		if (anyline.slope_value + 0.0000001 < slope)   // + 0.001 ��Ϊ�˷�ֹ��������
			return false;
		else if (max_intersection.pos_y > (((anyline.param_c - anyline.param_a * max_intersection.pos_x) / anyline.param_b) + 0.0000001))
			return false;
		else
			return true;
	}
};

struct fanzao
{
	const double x;
	fanzao(double _x) : x(_x) {}
	__host__ __device__
		double operator()(line anyline)
	{
		return (anyline.param_c - anyline.param_a * x) / anyline.param_b;
	}
};

struct one_plus_find_answer
{
	const line plus_line;
	one_plus_find_answer(line _plus_line) : plus_line(_plus_line) {}
	__host__ __device__
		double operator()(line minus_line)
	{
		return (plus_line.param_c * minus_line.param_a - plus_line.param_a * minus_line.param_c) / (plus_line.param_b * minus_line.param_a - plus_line.param_a * minus_line.param_b);
	}

};

void print_line(line * anyline)
{	
	std::cout << anyline->param_a << "x" << "+" << anyline->param_b << "y" << ">=" << anyline->param_c;
	std::cout << "   I_value" << anyline->I_value << "  slope" << anyline->slope_value << "  distance" << anyline->distance_with_testline;
	std::cout << std::endl;
}

void print_answer_by_me(answer_me ans)
{
	std::cout << "Answer is: " << ans.answer_b << std::endl;
	std::cout << "Line 1 is: " << ans.line1.param_a << "x +" << ans.line1.param_b << "y >=" << ans.line1.param_c << std::endl;
	std::cout << "Line 2 is: " << ans.line2.param_a << "x +" << ans.line2.param_b << "y >=" << ans.line2.param_c << std::endl;
	std::cout << "Intersection pointer is posX : " << ans.intersection_point.pos_x << ",   posY : " << ans.intersection_point.pos_y << std::endl;
}

void intersection_rotate_return(point &anypoint, double a, double b)
{
	double x = anypoint.pos_x;
	double y = anypoint.pos_y;
	double object_length = sqrt(a * a + b * b);
	anypoint.pos_x = (x * b + y * a) / object_length;
	anypoint.pos_y = (x * (-a) + y * b) / object_length;
}

void line_rotate_return(line &anyline, double a, double b)
{
	point node_x, node_y;
	node_x.pos_y = 0;
	node_x.pos_x = anyline.param_c / anyline.param_a;
	node_y.pos_x = 0;
	node_y.pos_y = anyline.param_c / anyline.param_b;
	intersection_rotate_return(node_x, a, b);
	intersection_rotate_return(node_y, a, b);
	bool z_0_befor = anyline.param_c > 0;
	anyline = *(generate_line_from_2points(&node_x, &node_y));
	bool z_0_after = anyline.param_c > 0;
	if (z_0_befor != z_0_after)
	{
		anyline.param_a = -anyline.param_a;
		anyline.param_b = -anyline.param_b;
		anyline.param_c = -anyline.param_c;
	}
}

void answer_rotate(answer_me&ans, double a, double b)
{
	line_rotate_return(ans.line1, a, b);
	line_rotate_return(ans.line2, a, b);
	intersection_rotate_return(ans.intersection_point, a, b);
}

int main()
{
	inputs * input = read_from_file(""); // replace this with your input files
	int num = input->number;
	line ** lines = input->lines;
	/*
	for (int i = 0; i < num; i++)
	{
		dev_lines.push_back(* lines[i]);
	}
	
	thrust::host_vector<line> host_lines(num);
	for (int i = 0; i < num; i++)
	{
		host_lines[i] = (*lines[i]);
	}
	*/
	thrust::device_vector<line> device_lines(num);
	for (int i = 0; i < num; i++)
	{
		device_lines[i] = (*lines[i]);
	}
	/*
	thrust::host_vector<line> host_lines(num);
	thrust::copy(device_lines.begin(), device_lines.end(), host_lines.begin());
	for (int i = 0; i < num; i++)
	{
		std::cout << host_lines[i].param_a << "\n" << std::endl;
	}
	*/
	double a, b;
	a = input->obj_function_param_a;
	b = input->obj_function_param_b;
	double object_length = sqrt(a*a + b*b);

	thrust::transform(device_lines.begin(), device_lines.end(), device_lines.begin(), rotate_functor(a, b));	
	thrust::host_vector<line> host_lines(num);
	/*
	thrust::copy(device_lines.begin(), device_lines.end(), host_lines.begin());
	for (int i = 0; i < num; i++)
	{
		std::cout << i << "   " << host_lines[i].param_a;
		std::cout << "   " << host_lines[i].param_b;
		std::cout << "   " << host_lines[i].param_c;
		std::cout << "   " << host_lines[i].I_value;
		std::cout << "\n" << std::endl;
	}
	*/


	// 7��26��ע��
	/*thrust::transform(device_lines.begin(), device_lines.end(), device_lines.begin(), generate_intersection(0));
	
	thrust::copy(device_lines.begin(), device_lines.end(), host_lines.begin());
	for (int i = 0; i < num; i++)
	{
		std::cout << i << "   " << host_lines[i].param_a;
		std::cout << "   " << host_lines[i].param_b;
		std::cout << "   " << host_lines[i].param_c;
		std::cout << "   " << host_lines[i].I_value;
		std::cout << "   " << host_lines[i].slope_value;
		std::cout << "   " << host_lines[i].distance_with_testline;
		std::cout << "\n" << std::endl;
	}*/



	//thrust::host_vector<intersection> host_lines_insertion(num);
	//thrust::copy(device_lines_insertion.begin(), device_lines_insertion.end(), host_lines_insertion.begin());

	//for (int i = 0; i < num; i++)
	//{
	//	std::cout << i << "   " << host_lines_insertion[i].pos_x;
	//	std::cout << "   " << host_lines_insertion[i].pos_y;
	//	std::cout << "\n" << std::endl;
	//}
	

	
	int number_of_Iminus = thrust::count_if(device_lines.begin(), device_lines.end(), is_I_minus());
	int number_of_Iplus = num - number_of_Iminus;
	//std::cout << number_of_Iminus << std::endl;
	thrust::device_vector<line> device_I_minus_lines(number_of_Iminus);
	thrust::device_vector<line> device_I_plus_lines(num - number_of_Iminus);
	thrust::partition_copy(device_lines.begin(), device_lines.end(), device_I_minus_lines.begin(), device_I_plus_lines.begin(), is_I_minus());
	//std::cout << device_I_minus_lines.size() << std::endl;
	//std::cout << device_I_plus_lines.size() << std::endl;
	answer_me ans;
	if (number_of_Iplus == 1)
	{
		line Iplus = device_I_plus_lines[0];
		thrust::device_vector<double> device_potential_answer(number_of_Iminus);
		thrust::host_vector<double> host_potential_answer(number_of_Iminus);
		thrust::transform(device_I_minus_lines.begin(), device_I_minus_lines.end(), device_potential_answer.begin(), one_plus_find_answer(Iplus));
		thrust::copy(device_potential_answer.begin(), device_potential_answer.end(), host_potential_answer.begin());
		double min = host_potential_answer[0];
		int minidx = 0;
		for (int i = 0; i < number_of_Iminus; i++)
		{
			if (host_potential_answer[i] < min)
			{
				min = host_potential_answer[i];
				minidx = i;
			}
		}
		ans.answer_b = min * object_length;
		ans.line1 = Iplus;
		ans.line2 = device_I_minus_lines[minidx];
		ans.intersection_point.pos_x = (Iplus.param_c * ans.line2.param_b - Iplus.param_b * ans.line2.param_c)
			/ (Iplus.param_a * ans.line2.param_b - Iplus.param_b * ans.line2.param_a);
		ans.intersection_point.pos_y = (Iplus.param_c * ans.line2.param_a - Iplus.param_a * ans.line2.param_c)
			/ (Iplus.param_b * ans.line2.param_a - Iplus.param_a * ans.line2.param_b);
	}
	if (number_of_Iplus > 2)
	{
		// ����Ϊɾ��I+ֱ�ߵĲ���

		double testline = 50;
		// ѡȡһ����ֱ��ֱ����Ϊ�ұ߽�,���ҵ��ұ߽��ϵ�����I+ֱ��
		line max_I_plus_line_rightbound;  // ���ұ߽�����I+ֱ��
		double rightbound;                    // �ұ߽�
		line max_I_plus_line;             // ����߽�����I+ֱ��
		double leftbound;                     // ��߽�

											  // ȷ��һ���ұ߽�
		while (true)
		{
			thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_I_plus_lines.begin(), generate_intersection(testline));
			auto o_0 = thrust::max_element(device_I_plus_lines.begin(), device_I_plus_lines.end(), compare_by_distance());
			max_I_plus_line_rightbound = device_I_plus_lines[o_0 - device_I_plus_lines.begin()];
			//std::cout << max_I_minus_line_rightbound.param_a << "  " << max_I_minus_line_rightbound.slope_value << std::endl;
			/*thrust::host_vector<line> host_I_minus_lines(number_of_Iminus);
			thrust::copy(device_I_minus_lines.begin(), device_I_minus_lines.end(), host_I_minus_lines.begin());*/
			/*for (int i = 0; i < number_of_Iminus; i++)
			{
			std::cout << i << "   " << host_I_minus_lines[i].param_a;
			std::cout << "   " << host_I_minus_lines[i].param_b;
			std::cout << "   " << host_I_minus_lines[i].param_c;
			std::cout << "   " << host_I_minus_lines[i].I_value;
			std::cout << "   " << host_I_minus_lines[i].slope_value;
			std::cout << "   " << host_I_minus_lines[i].distance_with_testline;
			std::cout << "\n" << std::endl;
			}
			break;*/
			if (max_I_plus_line_rightbound.slope_value > 0)
				break;
				//std::cout << max_I_plus_line_rightbound.slope_value << std::endl;
			testline = testline + 10;  // �����ֱ��Ϊ��߽磬������
		}
		rightbound = testline;
		//std::cout << "the rightbound is " << rightbound << std::endl;
		//print_line(&max_I_plus_line_rightbound);

		// �ұ߽�����ѡȡһ��ֱ�ߣ�������߽磬������I+ֱ��б��С��0����б�ʴ���0�����Ը�ֱ��Ϊ�ұ߽磬����Ѱ����߽�
		while (true)
		{
			testline = rightbound - 10;
			thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_I_plus_lines.begin(), generate_intersection(testline));
			auto o_0 = thrust::max_element(device_I_plus_lines.begin(), device_I_plus_lines.end(), compare_by_distance());
			max_I_plus_line = device_I_plus_lines[o_0 - device_I_plus_lines.begin()];
			//std::cout << max_I_minus_line.distance_with_testline << std::endl;
			if (max_I_plus_line.slope_value > 0)
			{
				rightbound = testline;
				//std::cout << "now the rightbound is " << rightbound << std::endl;
				max_I_plus_line_rightbound = max_I_plus_line;
				continue;
			}
			else
				leftbound = testline;
			break;
		}
		//std::cout << "the leftbound is " << leftbound << std::endl;
		//print_line(&max_I_plus_line);
		line for_move_bound;
		while (device_I_plus_lines.size() > 2)
		{
			// ������߽��ϵ����I+ֱ��
			thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_I_plus_lines.begin(), generate_intersection(leftbound));
			auto o_0 = thrust::max_element(device_I_plus_lines.begin(), device_I_plus_lines.end(), compare_by_distance());
			max_I_plus_line = device_I_plus_lines[o_0 - device_I_plus_lines.begin()];
			//print_line(&max_I_plus_line);

			/*thrust::host_vector<line> host_I_plus_lines(device_I_plus_lines.size());
			thrust::copy(device_I_plus_lines.begin(), device_I_plus_lines.end(), host_I_plus_lines.begin());
			std::cout << "the size of I plus line" << device_I_plus_lines.size() << std::endl;
			for (int i = 0; i < device_I_plus_lines.size(); i++)
			{
				std::cout << i << "   " << host_I_plus_lines[i].param_a;
				std::cout << "   " << host_I_plus_lines[i].param_b;
				std::cout << "   " << host_I_plus_lines[i].param_c;
				std::cout << "   " << host_I_plus_lines[i].I_value;
				std::cout << "   " << host_I_plus_lines[i].slope_value;
				std::cout << "   " << host_I_plus_lines[i].distance_with_testline;
				std::cout << "\n" << std::endl;
			}*/
			// �����ұ߽��ϵ����I+ֱ��
			thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_I_plus_lines.begin(), generate_intersection(rightbound));
			o_0 = thrust::max_element(device_I_plus_lines.begin(), device_I_plus_lines.end(), compare_by_distance());
			max_I_plus_line_rightbound = device_I_plus_lines[o_0 - device_I_plus_lines.begin()];
			//print_line(&max_I_plus_line_rightbound);

			////thrust::host_vector<line> host_I_plus_lines(device_I_plus_lines.size());
			//thrust::copy(device_I_plus_lines.begin(), device_I_plus_lines.end(), host_I_plus_lines.begin());
			//std::cout << "the size of I plus line" << device_I_plus_lines.size() << std::endl;
			//for (int i = 0; i < device_I_plus_lines.size(); i++)
			//{
			//	std::cout << i << "   " << host_I_plus_lines[i].param_a;
			//	std::cout << "   " << host_I_plus_lines[i].param_b;
			//	std::cout << "   " << host_I_plus_lines[i].param_c;
			//	std::cout << "   " << host_I_plus_lines[i].I_value;
			//	std::cout << "   " << host_I_plus_lines[i].slope_value;
			//	std::cout << "   " << host_I_plus_lines[i].distance_with_testline;
			//	std::cout << "\n" << std::endl;
			//}



			// �Ƴ�ֱ���Լ��ж����ұ߽����I+ֱ�ߵĽ��㴦�Ĵ�������߽绹���ұ߽�
			point * max_intersection = generate_intersection_point(&max_I_plus_line, &max_I_plus_line_rightbound);
			/*if (device_I_plus_lines.size() == 4)
			{
				std::cout << "intersection" << "(" << max_intersection->pos_x << "," << max_intersection->pos_y << std::endl;

				thrust::device_vector<double> device_fanzao(device_I_plus_lines.size());
				thrust::host_vector<double> host_fanzao(device_I_plus_lines.size());
				thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_fanzao.begin(), fanzao(max_intersection->pos_x));
				thrust::copy(device_fanzao.begin(), device_fanzao.end(), host_fanzao.begin());

				for (int i = 0; i < device_I_plus_lines.size(); i++)
				{
				std::cout << i << " " << host_fanzao[i] << std::endl;
				}
			}*/
			/*std::cout << "intersection" << "(" << max_intersection->pos_x << "," << max_intersection->pos_y << std::endl;

			thrust::device_vector<double> device_fanzao(device_I_plus_lines.size());
			thrust::host_vector<double> host_fanzao(device_I_plus_lines.size());
			thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_fanzao.begin(), fanzao(max_intersection->pos_x));
			thrust::copy(device_fanzao.begin(), device_fanzao.end(), host_fanzao.begin());

			for (int i = 0; i < device_I_plus_lines.size(); i++)
			{
				std::cout << i << " " << host_fanzao[i] << std::endl;
			}*/


			int no_del = thrust::count_if(device_I_plus_lines.begin(), device_I_plus_lines.end(), is_move_for_count(max_I_plus_line.slope_value, *max_intersection));
			thrust::remove_if(device_I_plus_lines.begin(), device_I_plus_lines.end(), is_move(max_I_plus_line.slope_value, *max_intersection));
			//std::cout << "after remove" << device_I_plus_lines.size() << std::endl;
			//std::cout << "no_del   " << no_del << std::endl;
			device_I_plus_lines.resize(no_del);
			//std::cout << "device_I_plus_lines" << device_I_plus_lines.size() << std::endl;
			testline = max_intersection->pos_x;
			thrust::transform(device_I_plus_lines.begin(), device_I_plus_lines.end(), device_I_plus_lines.begin(), generate_intersection(testline));
			o_0 = thrust::max_element(device_I_plus_lines.begin(), device_I_plus_lines.end(), compare_by_distance());
			for_move_bound = device_I_plus_lines[o_0 - device_I_plus_lines.begin()];
			if (for_move_bound.slope_value > 0)
				rightbound = testline;
			else
				leftbound = testline;
			////std::cout << "now the rightbound is " << rightbound << std::endl;
			//thrust::host_vector<line> host_I_plus_lines(device_I_plus_lines.size());
			//thrust::copy(device_I_plus_lines.begin(), device_I_plus_lines.end(), host_I_plus_lines.begin());
			//std::cout << "the size of I plus line" << device_I_plus_lines.size() << std::endl;
			//for (int i = 0; i < device_I_plus_lines.size(); i++)
			//{
			//	std::cout << i << "   " << host_I_plus_lines[i].param_a;
			//	std::cout << "   " << host_I_plus_lines[i].param_b;
			//	std::cout << "   " << host_I_plus_lines[i].param_c;
			//	std::cout << "   " << host_I_plus_lines[i].I_value;
			//	std::cout << "   " << host_I_plus_lines[i].distance_with_testline;
			//	std::cout << "\n" << std::endl;
			//}
		}
		number_of_Iplus = device_I_plus_lines.size();
	}
	if (number_of_Iplus == 2)
	{
		ans.line1 = device_I_plus_lines[0];
		ans.line2 = device_I_plus_lines[1];
		point * line1_line2 = generate_intersection_point(&ans.line1, &ans.line2);
		ans.answer_b = line1_line2->pos_y * object_length;
		ans.intersection_point = *line1_line2;
	}
	print_answer_by_me(ans);
	answer_rotate(ans, a, b);
	print_answer_by_me(ans);


	//thrust::host_vector<line> host_I_minus_lines(device_I_minus_lines.size());
	//thrust::copy(device_I_minus_lines.begin(), device_I_minus_lines.end(), host_I_minus_lines.begin());
	//for (int i = 0; i < device_I_minus_lines.size(); i++)
	//{
	//	std::cout << i << "   " << host_I_minus_lines[i].param_a;
	//	std::cout << "   " << host_I_minus_lines[i].param_b;
	//	std::cout << "   " << host_I_minus_lines[i].param_c;
	//	std::cout << "   " << host_I_minus_lines[i].I_value;
	//	std::cout << "   " << host_I_minus_lines[i].distance_with_testline;
	//	std::cout << "\n" << std::endl;
	//}
	
	


	/*
	thrust::device_vector<I> I_of_eachline(num);
	thrust::transform(device_lines.begin(), device_lines.end(), I_of_eachline.begin(), I_functor(0));
	thrust::host_vector<I> host_I_of_lines(num);
	thrust::copy(I_of_eachline.begin(), I_of_eachline.end(), host_I_of_lines.begin());
	*/
	/*
	for (int i = 0; i < num; i++)
	{
		std::cout << i << "   " << host_lines[i].param_a;
		std::cout << i << "   " << host_lines[i].param_b;
		std::cout << i << "   " << host_lines[i].param_c;
		std::cout << i << "   " << host_I_of_lines[i];
		std::cout << "\n" << std::endl;
	}
	*/
	//for (int i = 0; i < num; i++)
	//{
		//(*lines[i]) = host_lines[i];
		//std::cout << i << "   " << (*lines[i]).param_a << std::endl;
	//}
	//inputs * rotate_input = (inputs *)malloc(sizeof(inputs));;
	//rotate_input->lines = lines;
	//for (int i = 0; i < num; i++)
	//{
		//(*lines[i]) = host_lines[i];
	//	std::cout << i << "   " << (*rotate_input->lines[i]).param_a << std::endl;
	//}
	//rotate_input->obj_function_param_a = 0;
	//rotate_input->obj_function_param_b = sqrt(a*a+b*b);
	/*
	answer * ans = compute(input);
	rotate_all(input);
	for (int i = 0; i < num; i++)
	{
		std::cout << i << "   " << (*input->lines[i]).param_a;
		std::cout << i << "   " << (*input->lines[i]).param_b;
		std::cout << i << "   " << (*input->lines[i]).param_c;
		std::cout << "\n" << std::endl;
	}
	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	free_inputs(&input);
	//free_inputs(&rotate_input);
	//free_inputs(&rotate_input);
	free_ans(&ans);
	free(ans_string);
	*/
	return 0;
}