#include "hip/hip_runtime.h"
#include "floating_number_helper.h"
#include "input_output.h"
#include <float.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include <iostream>
#include ""

#include "PerformanceTimer.h"

PerformanceTimer timer;
#define THREAD_NUM 256

//
// this function is what you need to finish
// @Usage : to solve the problem
// @Input : input containing all data needed
// @Output: answer containing all necessary data
//  you can find the definition of the two structs above in
//      input_output.h
//

answer * compute(inputs * input) {
	int num = input->number;
	line ** lines = input->lines;
	answer * ans = (answer *)malloc(sizeof(answer));
	double tmp = 0;
	//double min = MAXFLOAT;
	double min = FLT_MAX;
	for (int i = 0; i < num - 1; i++) {
		line * now_line = lines[i];
		for (int j = i + 1; j < num; j++) {
			line * tmp_line = lines[j];
			point * tmp_node = generate_intersection_point(now_line, tmp_line);
			//std::cout << tmp_node->pos_x << "   " << tmp_node->pos_y << std::endl;
			tmp = input->obj_function_param_a * tmp_node->pos_x + input->obj_function_param_b * tmp_node->pos_y;
			//std::cout << tmp << std::endl;
			if (tmp <= min) {
				for (int k = 0; k < num; k++) {
					if ((tmp_node->pos_x * lines[k]->param_a + tmp_node->pos_y * lines[k]->param_b) < lines[k]->param_c) {
						break;
					}
					else if (k == num - 1) {
						//std::cout << "we can get answer" << std::endl;
						min = tmp;
						ans->answer_b = tmp;
						ans->intersection_point = tmp_node;
						ans->line1 = now_line;
						ans->line2 = tmp_line;
					}
				}
			}
		}
	}
	return ans;
}

__device__ int equals_gpu(double num1, double num2) {
	return fabs(num1 - num2) < EPS ? TRUE : FALSE;
}

__device__ int is_parallel_gpu(line * line1, line * line2) {
	return equals_gpu(line1->param_a * line2->param_b, line1->param_b * line2->param_a);
}

__device__ point * generate_intersection_point_gpu(line * line1, line * line2) {
	if (is_parallel_gpu(line1, line2)) {
		return NULL;
	}
	point * new_point = (point *)malloc(sizeof(point));
	new_point->pos_x = (line1->param_c * line2->param_b - line1->param_b * line2->param_c)
		/ (line1->param_a * line2->param_b - line1->param_b * line2->param_a);
	new_point->pos_y = (line1->param_c * line2->param_a - line1->param_a * line2->param_c)
		/ (line1->param_b * line2->param_a - line1->param_a * line2->param_b);
	return new_point;
}

__global__ static void gpu_compute(int * dev_num, double * dev_param_a, 
	         double * dev_param_b, line * dev_line_array, for_answer * dev_potential) //, double* arr) 
{
	int num = *dev_num;
	double a = *dev_param_a;
	double b = *dev_param_b;
	double min = FLT_MAX;
	int line2 = 0 ;
	//int i = threadIdx.x; 
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < num)
	{
		for (int j = i + 1; j < num; j++)
		{
			point * tmp_node = generate_intersection_point_gpu(&dev_line_array[i], &dev_line_array[j]);
			double tmp = a * tmp_node->pos_x + b * tmp_node->pos_y;
			//printf("%d  %d  %f\n", i, j, tmp);
			for (int k = 0; k < num; k++)
			{
				if ((tmp_node->pos_x * dev_line_array[k].param_a + tmp_node->pos_y * dev_line_array[k].param_b) < dev_line_array[k].param_c - 0.0001)
				{
					//printf("%f", tmp);
					break;
				}
				else if (k == num - 1 && tmp <= min)
				{
					min = tmp;
					line2 = j;
				}
			}
		}
		 //printf("%d  %d  %f\n", i, line2, min);
		dev_potential[i].line1 = threadIdx.x;
		dev_potential[i].line2 = line2;
		dev_potential[i].possible = min;
	}

}

void checkCUDAError() {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
	}
}

int main() {
	hipError_t err = hipSuccess;

	// 1. get the input data
	inputs * input = read_from_file("C:/Users/hzauz/Desktop/sws3003_assignment 1/test_cases/1_0.dat");
	int num = input->number;
	
	for_answer * potential = (for_answer*)malloc(sizeof(for_answer) * num);
	line * line_array = (line*)malloc(sizeof(line) * num);
	if (line_array == NULL || potential == NULL)
	{
		fprintf(stderr, "Failed to allocate host input data!\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < num; i++)
	{
		potential[i].possible = FLT_MAX;
		// debug
		potential[i].line1 = i;
	}
                            
	for (int i = 0; i < num; i++)
	{
		line_array[i] = *((input->lines)[i]);
	}

	for_answer * dev_potential = NULL;
	err = hipMalloc((void**)&dev_potential, sizeof(for_answer) * num);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device potential vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(dev_potential, potential, sizeof(for_answer) * num, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy potential vector from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	line * dev_line_array = NULL;
	err = hipMalloc((void**)&dev_line_array, sizeof(line) * num);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device line (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	err = hipMemcpy(dev_line_array, line_array, sizeof(line) * num, hipMemcpyHostToDevice);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy line array from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	
	int * dev_num;                                                                      // for input the number of line
	hipMalloc((void**)&dev_num, sizeof(int));
	hipMemcpy(dev_num, &num, sizeof(int), hipMemcpyHostToDevice);

	double * dev_param_a;                                                               // for input the number of a and b
	double param_a = input->obj_function_param_a;
	hipMalloc((void**)&dev_param_a, sizeof(double));
	hipMemcpy(dev_param_a, &param_a, sizeof(double), hipMemcpyHostToDevice);

	double * dev_param_b;
	double param_b = input->obj_function_param_b;
	hipMalloc((void**)&dev_param_b, sizeof(double));
	hipMemcpy(dev_param_b, &param_b, sizeof(double), hipMemcpyHostToDevice);
	
	// 3.computing by using gpu
	int blocks_num = (num + THREAD_NUM - 1) / THREAD_NUM;
	//gpu_compute<<<blocks_num,THREAD_NUM>>>(dev_num, dev_param_a, dev_param_b, dev_line_array, dev_potential);
	int threadsPerBlock = 256;
	int blocksPerGrid = (num + threadsPerBlock - 1) / threadsPerBlock;
	printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	timer.StartTimer();
	gpu_compute << <blocksPerGrid, threadsPerBlock >> >(dev_num, dev_param_a, dev_param_b, dev_line_array, dev_potential);
	printf("Processing time (GPU): %f (ms) \n", timer.GetTimeElapsed() * 1000);
	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch Linear programmer kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}



	// 4.getting the final answer
	err = hipMemcpy(potential, dev_potential, sizeof(for_answer) * num, hipMemcpyDeviceToHost);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy potential from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	double min = FLT_MAX;
	int minidx = 0;
	for (int i = 0; i < num; i++)
	{
		//std::cout << potential[i].line1 << "   " << potential[i].line2 << std::endl;
		if (potential[i].possible < min - 0.0001)
		{
			min = potential[i].possible;
			minidx = i;
		}
	}
	
	//std::cout << "the minidx is :" << minidx << std::endl;
	//std::cout << potential[minidx].line1 << " " << potential[minidx].line2 << std::endl;

	if (minidx == 0 && min - FLT_MAX < 0.1 && min - FLT_MAX > -0.1)
	{
		printf("Failed to calculate the answer.\n");
		exit(EXIT_FAILURE);
	}

	answer * ans = (answer *)malloc(sizeof(answer));
	point * tmp_node = generate_intersection_point(&line_array[potential[minidx].line1], &line_array[potential[minidx].line2]);
	if (tmp_node == NULL)
	{
		std::cout << "No resolution" << std::endl;
		return 0;
	}
	
	
	double tmp = param_a * tmp_node->pos_x + param_b * tmp_node->pos_y;
	ans->answer_b = tmp;
	ans->intersection_point = tmp_node;
	ans->line1 = input->lines[potential[minidx].line1];
	ans->line2 = input->lines[potential[minidx].line2];
	// 3. display result and free memory
	char * ans_string = generate_ans_string(ans);
	printf("%s", ans_string);
	free_inputs(&input);
	hipFree(dev_line_array);
	free(line_array);
	free_ans(&ans);
	free(ans_string);
	hipFree(dev_potential);
	free(potential);

	return 0;
}

