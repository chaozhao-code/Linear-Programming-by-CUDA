//
// Created by 唐艺峰 on 2018/7/14.
//

#include "input_output.h"

inputs * read_from_file(char * filename) {
    inputs * input = (inputs *) malloc(sizeof(inputs));
    FILE * input_file = fopen(filename, "r");
    if (input_file == NULL) {
        printf("Cannot open the file.");
        exit(1);
    }
    int num_of_lines;
    fscanf(input_file, "%d", &num_of_lines);
    line ** lines = (line **) malloc(sizeof(line *) * num_of_lines);
    for (int line_no = 0; line_no < num_of_lines; line_no++) {
        double param_a, param_b, param_c;
        fscanf(input_file, "%lf %lf %lf", &param_a, &param_b, &param_c);
        line * new_line = generate_line_from_abc(param_a, param_b, param_c);
        lines[line_no] = new_line;
    }
    fscanf(input_file, "%lf %lf", &input->obj_function_param_a, &input->obj_function_param_b);
    input->lines = lines;
    input->number = num_of_lines;
    if (fclose(input_file) != 0) {
        printf("Error for closing this files");
        exit(0);
    }
    return input;
}

char * generate_ans_string(answer * ans) {
    char * ans_string = (char *) malloc(sizeof(char) * DEFAULT_ANS_LEN);
    sprintf(ans_string, "Answer is: %lf\n"
                    "Line 1 is : %lfx + %lfy >= %lf\n"
                    "Line 2 is : %lfx + %lfy >= %lf\n"
                    "Intersection point is (%lf, %lf)\n",
            ans->answer_b,
            ans->line1->param_a, ans->line1->param_b, ans->line1->param_c,
            ans->line2->param_a, ans->line2->param_b, ans->line2->param_c,
            ans->intersection_point->pos_x, ans->intersection_point->pos_y);
    return ans_string;
}

void free_inputs(inputs ** input) {
    inputs * aim = * input;
    for (int i = 0; i < aim->number; i++) {
        free(aim->lines[i]);
    }
    free(aim);
}

void free_ans(answer ** ans) {
    answer * aim = * ans;
//    free(aim->line1);
//    free(aim->line2);
//    take care of double-free!
    free(aim->intersection_point);
    free(aim);
}

